
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstring>
#include <fstream>
#include "time.h"

using namespace std;

void preKMP(char* pattern, int f[])
{
    int m = strlen(pattern), k;
    f[0] = -1;
    for (int i = 1; i < m; i++)
    {
        k = f[i - 1];
        while (k >= 0)
        {
            if (pattern[k] == pattern[i - 1])
                break;
            else
                k = f[k];
        }
        f[i] = k + 1;
    }
}
 
//check whether target string contains pattern 
__global__ void KMP(char* pattern, char* target,int f[],int c[],int n, int m)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    int i = n * index;
    int j = n * (index + 2)-1;
    if(i>m)
        return;
    if(j>m)
        j=m;
    int k = 0;        
    while (i < j)
    {
        if (k == -1)
        {
            i++;
            k = 0;
        }
        else if (target[i] == pattern[k])
        {
            i++;
            k++;
            if (k == n)
            {
                c[i - n] = i-n;
                i = i - k + 1;
            }
        }
        else
            k = f[k];
    }
    return;
}
 
int main(int argc, char* argv[])
{
    const int L = 40000000;
    const int S = 40000000;
    int M = 1024;//num of threads

    int cSize = 4;//size of char is 1, but size of 'a' is 4

    char *tar;
    char *pat;
    tar = (char*)malloc(L*cSize);
    pat = (char*)malloc(S*cSize);
    char *d_tar;
    char *d_pat;
    ifstream f1;
    ofstream f2;

    f1.open(argv[1]);
    f2.open("output.txt");

    f1>>tar>>pat;

    int m = strlen(tar);
    int n = strlen(pat);
    printf("%d %d\n",m,n);
    int *f;
    int *c;

    f = new int[m];
    c = new int[m];

    int *d_f;
    int *d_c;
    for(int i = 0;i<m; i++)
    {
        c[i] = -1;
    }     
    preKMP(pat, f);
    printf("----Start copying data to GPU----\n");
    time_t rawtime1;
    time ( &rawtime1 );
    hipMalloc((void **)&d_tar, m*cSize);
    hipMalloc((void **)&d_pat, n*cSize);
    hipMalloc((void **)&d_f, m*cSize);
    hipMalloc((void **)&d_c, m*cSize);

    hipMemcpy(d_tar, tar, m*cSize, hipMemcpyHostToDevice);
    hipMemcpy(d_pat, pat, n*cSize, hipMemcpyHostToDevice);
    hipMemcpy(d_f, f, m*cSize, hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, m*cSize, hipMemcpyHostToDevice);
    time_t rawtime2;
    time ( &rawtime2 );
    printf("----Data copied to GPU successfully---- Takes %f seconds\n", difftime(rawtime2,rawtime1));
    if(n>10000000)
        M = 128;

//使用event计算时间
    float time_elapsed=0;
    hipEvent_t start,stop;
    hipEventCreate(&start);    //创建Event
    hipEventCreate(&stop);

    hipEventRecord( start,0);    //记录当前时间
    KMP<<<(m/n+M)/M,M>>>(d_pat, d_tar ,d_f, d_c, n, m);
    hipEventRecord( stop,0);    //记录当前时间
 
    hipEventSynchronize(start);    //Waits for an event to complete.
    hipEventSynchronize(stop);    //Waits for an event to complete.Record之前的任务
    hipEventElapsedTime(&time_elapsed,start,stop);    //计算时间差


    printf("----String matching done---- Takes %f s\n", time_elapsed/1000);  
    
    hipMemcpy(c, d_c, m*cSize, hipMemcpyDeviceToHost);

    for(int i = 0;i<m; i++)
    { 
        if(c[i]!=-1)
        {
            f2<<i<<' '<<c[i]<<'\n';
        }
    }
    time_t rawtime4;
    time ( &rawtime4 );
    printf("----Task done---- Takes %f seconds in total\n", difftime(rawtime4,rawtime1));
    hipFree(d_tar); hipFree(d_pat); hipFree(d_f); hipFree(d_c);
    return 0;
}