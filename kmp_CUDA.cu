
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstring>
#include <fstream>

using namespace std;

void preKMP(char* pattern, int f[])
{
    int m = strlen(pattern), k;
    f[0] = -1;
    for (int i = 1; i < m; i++)
    {
        k = f[i - 1];
        while (k >= 0)
        {
            if (pattern[k] == pattern[i - 1])
                break;
            else
                k = f[k];
        }
        f[i] = k + 1;
    }
}
 
//check whether target string contains pattern 
__global__ void KMP(char* pattern, char* target,int f[],int c[],int m)
{

    int i = m * blockIdx.x;
    int n = m * (blockIdx.x + 2)-1;
    int k = 0;        
    while (i < n)
    {
        if (k == -1)
        {
            i++;
            k = 0;
        }
        else if (target[i] == pattern[k])
        {
            i++;
            k++;
            if (k == m)
            {
                c[i - m] = i-m;
            }
        }
        else
            k = f[k];
    }
    return;
}
 
int main(int argc, char* argv[])
{
    const int L = 40000000;
    const int S = 4000;
    const int N = 40000;// num of blocks

    int cSize = 4;//size of char is 1, but size of 'a' is 4

    char *tar;
    char *pat;
    tar = (char*)malloc(L*cSize);
    pat = (char*)malloc(S*cSize);
    char *d_tar;
    char *d_pat;
    ifstream f1;
    ofstream f2;

    f1.open(argv[1]);
    f2.open("output.txt");

    f1>>tar>>pat;

    int m = strlen(tar);
    int n = strlen(pat);
    int *f;
    int *c;
    printf("5\n");
    f = new int[m];
    c = new int[m];
    printf("6\n");
    int *d_f;
    int *d_c;
    for(int i = 0;i<m; i++)
    {
        c[i] = -1;
    }     
    preKMP(pat, f);
    printf("6\n");
    hipMalloc((void **)&d_tar, m*cSize);
    hipMalloc((void **)&d_pat, n*cSize);
    hipMalloc((void **)&d_f, m*cSize);
    hipMalloc((void **)&d_c, m*cSize);

    hipMemcpy(d_tar, tar, m*cSize, hipMemcpyHostToDevice);
    hipMemcpy(d_pat, pat, n*cSize, hipMemcpyHostToDevice);
    hipMemcpy(d_f, f, m*cSize, hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, m*cSize, hipMemcpyHostToDevice);
    printf("6\n");
    KMP<<<m/n,1>>>(d_pat, d_tar ,d_f, d_c, n);

    hipMemcpy(c, d_c, m*cSize, hipMemcpyDeviceToHost);

    for(int i = 0;i<m; i++)
    { 
        if(c[i]!=-1)
        {
            f2<<i<<' '<<c[i]<<'\n';
        }
    }

    hipFree(d_tar); hipFree(d_pat); hipFree(d_f); hipFree(d_c);
    return 0;
}