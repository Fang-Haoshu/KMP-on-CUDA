
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstring>
#include <fstream>

using namespace std;

void preKMP(char* pattern, int f[])
{
    int m = strlen(pattern), k;
    f[0] = -1;
    for (int i = 1; i < m; i++)
    {
        k = f[i - 1];
        while (k >= 0)
        {
            if (pattern[k] == pattern[i - 1])
                break;
            else
                k = f[k];
        }
        f[i] = k + 1;
    }
}
 
//check whether target string contains pattern 
__global__ void KMP(char* pattern, char* target,int f[],int c[],int n, int m)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if(index > m-n)
        return;
    int i = index;
    int j = index + n;
    int k = 0;        
    while (i < j)
    {
        if (k == -1)
        {
            i++;
            k = 0;
        }
        else if (target[i] == pattern[k])
        {
            i++;
            k++;
            if (k == n)
            {
                c[i - n] = i-n;
            }
        }
        else
            k = f[k];
    }
    return;
}
 
int main(int argc, char* argv[])
{
    const int L = 40000000;
    const int S = 4000;
    const int N = 40000;// num of blocks
    const int M = 1024;//num of threads

    int cSize = 4;//size of char is 1, but size of 'a' is 4

    char *tar;
    char *pat;
    tar = (char*)malloc(L*cSize);
    pat = (char*)malloc(S*cSize);

    ifstream f1;
    ofstream f2;

    f1.open(argv[1]);
    f2.open("output.txt");

    f1>>tar>>pat;

    int m = strlen(tar);
    int n = strlen(pat);
    int *f;
    int *c;
    f = new int[m];
    c = new int[m];
    for(int i = 0;i<m; i++)
    {
        c[i] = -1;
    }   

    preKMP(pat, f);

    char *d_tar;
    char *d_pat;
    int *d_f;
    int *d_c;

    hipMalloc((void **)&d_tar, m*cSize);
    hipMalloc((void **)&d_pat, n*cSize);
    hipMalloc((void **)&d_f, m*cSize);
    hipMalloc((void **)&d_c, m*cSize);

    hipMemcpy(d_tar, tar, m*cSize, hipMemcpyHostToDevice);
    hipMemcpy(d_pat, pat, n*cSize, hipMemcpyHostToDevice);
    hipMemcpy(d_f, f, m*cSize, hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, m*cSize, hipMemcpyHostToDevice);

    KMP<<<(m+M-1)/M,M>>>(d_pat, d_tar ,d_f, d_c, n, m);

    hipMemcpy(c, d_c, m*cSize, hipMemcpyDeviceToHost);

    for(int i = 0;i<m; i++)
    { 
        if(c[i]!=-1)
        {
            f2<<i<<' '<<c[i]<<'\n';
        }
    }

    hipFree(d_tar); hipFree(d_pat); hipFree(d_f); hipFree(d_c);
    return 0;
}