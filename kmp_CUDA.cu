
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstring>
#include <fstream>

using namespace std;

void preKMP(char* pattern, int f[])
{
    int m = strlen(pattern), k;
    f[0] = -1;
    for (int i = 1; i < m; i++)
    {
        k = f[i - 1];
        while (k >= 0)
        {
            if (pattern[k] == pattern[i - 1])
                break;
            else
                k = f[k];
        }
        f[i] = k + 1;
    }
}
 
//check whether target string contains pattern 
__global__ void KMP(char* pattern, char* target,int f[],int c[],int m)
{

    int i = 1000 * blockIdx.x;
    int n = 1000 * (blockIdx.x + 2);
    int k = 0;        
    while (i < n)
    {
        if (k == -1)
        {
            i++;
            k = 0;
        }
        else if (target[i] == pattern[k])
        {
            i++;
            k++;
            if (k == m)
            {
                c[blockIdx.x] = i-m;
                return;
            }
        }
        else
            k = f[k];
    }
    return;
}
 
int main(int argc, char* argv[])
{
    const int L = 40000000;
    const int S = 4000;
    const int N = 40000;// num of blocks
    char tar[L];
    char pat[S];
    char *d_tar;
    char *d_pat;
    ifstream f1;
    ofstream f2;
    f1.open(argv[0]);
    f2.open("output.txt");

    f1>>tar>>pat;

    int m = strlen(tar);
    int n = strlen(pat);
    int f[m];
    int c[N];
    int *d_f;
    int *d_c;
    for(int i = 0;i<N; i++)
    {
        c[i] = -1;
    }     
    preKMP(pat, f);

    hipMalloc((void **)&d_tar, L);
    hipMalloc((void **)&d_pat, S);
    hipMalloc((void **)&d_f, m);
    hipMalloc((void **)&d_c, N);

    hipMemcpy(d_tar, tar, L, hipMemcpyHostToDevice);
    hipMemcpy(d_pat, pat, S, hipMemcpyHostToDevice);
    hipMemcpy(d_f, f, m, hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, N, hipMemcpyHostToDevice);
    KMP<<<N,1>>>(d_pat, d_tar ,d_f, d_c, n);

    hipMemcpy(c, d_c, N, hipMemcpyDeviceToHost);

    for(int i = 0;i<N; i++)
    {
        if(c[i]!=-1)
        {
            f2<<c[i]<<'\n';
        }
    }

    hipFree(d_tar); hipFree(d_pat); hipFree(d_f); hipFree(d_c);
    return 0;
}