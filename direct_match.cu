
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstring>
#include <fstream>

using namespace std;

 
//check whether target string contains pattern 
void strMatch(char* pattern, char* target,int c[],int n, int m)
{
    for(int i = 0;i<m-n;i++)
    {
    int k = 0;        
    while (k < n)
    {
        if (pattern[k] == target[i+k])
        {
            k++;
        }
        else
           continue;
    }
    c[i] = i;
    }
    return;
}
 
int main(int argc, char* argv[])
{
    const int L = 40000000;
    const int S = 40000000;
    const int N = 40000;// num of blocks
    const int M = 1024;//num of threads

    int cSize = 4;//size of char is 1, but size of 'a' is 4

    char *tar;
    char *pat;
    tar = new char[L];
    pat = new char[S];

    ifstream f1;
    ofstream f2;

    f1.open(argv[1]);
    f2.open("output.txt");

    f1>>tar>>pat;

    int m = strlen(tar);
    int n = strlen(pat);
    int *c;
    c = new int[m];
    for(int i = 0;i<m; i++)
    {
        c[i] = -1;
    }   

    clock_t start,end;
    start = clock();
    strMatch(pat, tar,c, n, m);
    end = clock();

    printf("----String matching done---- Takes %f s\n", (end - start)/1000);  

    for(int i = 0;i<m; i++)
    { 
        if(c[i]!=-1)
        {
            f2<<i<<' '<<c[i]<<'\n';
        }
    }
    
    return 0;
}