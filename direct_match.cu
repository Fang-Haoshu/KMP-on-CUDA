
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstring>
#include <fstream>

using namespace std;

 
//check whether target string contains pattern 
__global__ void strMatch(char* pattern, char* target,int c[],int n, int m)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if(index > m-n)
        return;
    int i = index;
    int k = 0;        
    while (k < n)
    {
        if (pattern[k] == target[i+k])
        {
            k++;
        }
        else
           return;
    }
    c[i] = i;
    return;
}
 
int main(int argc, char* argv[])
{
    const int L = 40000000;
    const int S = 40000000;
    const int N = 40000;// num of blocks
    const int M = 1024;//num of threads

    int cSize = 4;//size of char is 1, but size of 'a' is 4

    char *tar;
    char *pat;
    tar = (char*)malloc(L*cSize);
    pat = (char*)malloc(S*cSize);

    ifstream f1;
    ofstream f2;

    f1.open(argv[1]);
    f2.open("output.txt");

    f1>>tar>>pat;

    int m = strlen(tar);
    int n = strlen(pat);
    int *c;
    c = new int[m];
    for(int i = 0;i<m; i++)
    {
        c[i] = -1;
    }   

    char *d_tar;
    char *d_pat;
    int *d_c;

    hipMalloc((void **)&d_tar, m*cSize);
    hipMalloc((void **)&d_pat, n*cSize);
    hipMalloc((void **)&d_c, m*cSize);

    hipMemcpy(d_tar, tar, m*cSize, hipMemcpyHostToDevice);
    hipMemcpy(d_pat, pat, n*cSize, hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, m*cSize, hipMemcpyHostToDevice);

    strMatch<<<(m+M-1)/M,M>>>(d_pat, d_tar , d_c, n, m);

    hipMemcpy(c, d_c, m*cSize, hipMemcpyDeviceToHost);

    for(int i = 0;i<m; i++)
    { 
        if(c[i]!=-1)
        {
            f2<<i<<' '<<c[i]<<'\n';
        }
    }

    hipFree(d_tar); hipFree(d_pat);  hipFree(d_c);
    return 0;
}